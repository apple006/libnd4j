#include "hip/hip_runtime.h"
//
// Created by GS <sgazeos@gmail.com> on 16.04.18.
//

#include <ops/declarable/helpers/matmul.h>
#define MAX_CUDA_DEVICE 512
#define BLOCK_SIZE 32
namespace nd4j {
    namespace ops {
        namespace helpers {
            
            //(CblasColMajor, transA, transB, M, N, K, (float) alpha, (float *) pA->getBuffer(), lda, (float *) pB->getBuffer(), ldb, (float) beta, (float *) pC->getBuffer(), ldc)
            template <typename T>
            void GUDA_G device_matmul(int align, int transA, int transB, int M, int N, int K, T alpha, T* pA, int ldA, T* pB, int ldB, T beta, T* pC, int ldC) {
                // Handle to thread block group
                cooperative_groups::thread_block cta = cooperative_groups::this_thread_block();
                // Block index
                int bx = blockIdx.x;
                int by = blockIdx.y;
    
                // Thread index
                int tx = threadIdx.x;
                int ty = threadIdx.y;
    
                // Index of the first sub-matrix of A processed by the block
                int aBegin = K * BLOCK_SIZE * by;
    
                // Index of the last sub-matrix of A processed by the block
                int aEnd   = aBegin + K - 1;
    
                // Step size used to iterate through the sub-matrices of A
                int aStep  = BLOCK_SIZE;
    
                // Index of the first sub-matrix of B processed by the block
                int bBegin = BLOCK_SIZE * bx;
    
                // Step size used to iterate through the sub-matrices of B
                int bStep  = BLOCK_SIZE * N;
    
                // Csub is used to store the element of the block sub-matrix
                // that is computed by the thread
                float Csub = 0;
    
                // Loop over all the sub-matrices of A and B
                // required to compute the block sub-matrix
                for (int a = aBegin, b = bBegin; a <= aEnd; a += aStep, b += bStep)
                {
                    // Declaration of the shared memory array As used to
                    // store the sub-matrix of A
                    __shared__ T As[BLOCK_SIZE][BLOCK_SIZE];
    
                    // Declaration of the shared memory array Bs used to
                    // store the sub-matrix of B
                    __shared__ T Bs[BLOCK_SIZE][BLOCK_SIZE];
    
                    // Load the matrices from device memory
                    // to shared memory; each thread loads
                    // one element of each matrix
                    As[ty][tx] = pA[a + K * ty + tx];
                    Bs[ty][tx] = pB[b + N * ty + tx];
    
                    // Synchronize to make sure the matrices are loaded
                   cooperative_groups::sync(cta);
    
    
                    // Multiply the two matrices together;
                    // each thread computes one element
                    // of the block sub-matrix
#pragma unroll
                    for (int k = 0; k < BLOCK_SIZE; ++k)
                    {
                        Csub += As[ty][k] * Bs[k][tx];
                    }
    
                    // Synchronize to make sure that the preceding
                    // computation is done before loading two new
                    // sub-matrices of A and B in the next iteration
                    cooperative_groups::sync(cta);
                }

                // Write the block sub-matrix to device memory;
                // each thread writes one element
                int c = N * BLOCK_SIZE * by + BLOCK_SIZE * bx;
                pC[c + N * ty + tx] = Csub;
            }

            template <typename T> 
            _CUDA_H
            void _matmul(NDArray<T> *vA, NDArray<T> *vB, NDArray<T> *vC, int transA, int transB, T alpha, T beta) {

                int M = vA->sizeAt(0);
                int N = vB->sizeAt(1);
                int K = vA->sizeAt(1);
                bool transAFlag = transA != CblasNoTrans;
                bool transBFlag = transB != CblasNoTrans;
                int ldA = transAFlag ? K : M;
                int ldB = transBFlag ? N : K;
                int ldC = M;
            
                device_matmul<T><<<MAX_CUDA_DEVICE, MAX_CUDA_DEVICE>>>(CblasColMajor, transA, transB, M, N, K, alpha, vA->getBuffer(), lda, vB->getBuffer(), ldb, beta, vC->getBuffer(), ldC);
            }

            template void _matmul<float>(NDArray<float> *A, NDArray<float> *B, NDArray<float> *C, int transA, int transB, float alpha, float beta);
            template void _matmul<float16>(NDArray<float16> *A, NDArray<float16> *B, NDArray<float16> *C, int transA, int transB, float16 alpha, float16 beta);
            template void _matmul<double>(NDArray<double> *A, NDArray<double> *B, NDArray<double> *C, int transA, int transB, double alpha, double beta);
        }
    }
}
